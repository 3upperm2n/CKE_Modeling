#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// 80 x 80 
const int N = 80 * 80; 

// 160 x 160 
//const int N = 160 * 160; 

// 320 x 320 
//const int N = 320 * 320; 

// 640 x 640 
//const int N = 640 * 640; 


// 1k x 4
//const int N = 1000; 

// 10k x 4
//const int N = 1000 * 10; 

// 100k x 4
//const int N = 1000 * 100;

// 1M x 4
//const int N = 1000 * 1000;

// 10M x 4
//const int N = 1000 * 1000 * 10;


//const int N = 1 << 20;

#define FLTSIZE sizeof(float)

inline int BLK(int data, int blocksize)
{
	return (data + blocksize - 1) / blocksize;
}

__global__ void kernel_vectorAdd (const float* __restrict__ a_d, 
		const float* __restrict__ b_d,
		const int N,
		const int offset,
		float *c_d)
{
	int tid = threadIdx.x + __mul24(blockIdx.x, blockDim.x);

	if(tid < N) {
		c_d[tid + offset] = a_d[tid + offset] + b_d[tid + offset];	
	}
}

int main( int argc, char **argv)
{
	int devid = 0 ;

	int num_streams = 1;

	if(argc >= 2)
		devid = atoi(argv[1]);

	hipSetDevice(devid);
/*
	printf("\nrunning %d cuda streams on device %d\n", num_streams, devid);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	printf("Device Number: %d\n", devid);
	printf("  Device name: %s\n", prop.name);
	printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
	printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	printf("  Concurrent copy and execution: %s\n",  (prop.deviceOverlap ? "Yes" : "No"));
	printf("  Concurrent kernels: %d\n",  (prop.concurrentKernels));
	printf("  Concurrent copy and kernel execution: %s with %d copy engine(s)\n", 
			(prop.deviceOverlap ? "Yes" : "No"), prop.asyncEngineCount);
*/


	//------------------------------------------------------------------------//
	// allocate data on the host
	//------------------------------------------------------------------------//
	size_t databytes = N  * FLTSIZE; 

	//float *a_h = (float*) malloc ( N * num_streams * FLTSIZE);
	//float *b_h = (float*) malloc ( N * num_streams * FLTSIZE);
	//float *c_h = (float*) malloc ( N * num_streams * FLTSIZE);

	float *a_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&a_h, N * num_streams * FLTSIZE));

	float *b_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&b_h, N * num_streams * FLTSIZE));

	float *c_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&c_h, N * num_streams * FLTSIZE));

	for(int i=0; i< N * num_streams; i++) {
		a_h[i] = 1.1f;	
		b_h[i] = 2.2f;	
	}

	//------------------------------------------------------------------------//
	// allocate data on the device 
	//------------------------------------------------------------------------//
	float *a_d;
	float *b_d;
	float *c_d;
	hipMalloc((void**)&a_d, N * num_streams * FLTSIZE);
	hipMalloc((void**)&b_d, N * num_streams * FLTSIZE);
	hipMalloc((void**)&c_d, N * num_streams * FLTSIZE);

	// kernel configuration
	dim3 threads = dim3(256, 1, 1);
	dim3 blocks  = dim3(BLK(N, threads.x), 1, 1);

	// create cuda event handles
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	hipEventRecord(start,0);

	// copy data to deivce
	hipMemcpyAsync(&a_d[0], &a_h[0],  databytes, hipMemcpyHostToDevice);
	hipMemcpyAsync(&b_d[0], &b_h[0],  databytes, hipMemcpyHostToDevice);

	// launch one worker kernel per stream
	kernel_vectorAdd <<< blocks, threads >>> (a_d, 
			b_d, 
			N, 
			0,
			c_d);

	// copy data back to host
	hipMemcpyAsync(&c_h[0], &c_d[0],  databytes, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);

	// required for async copy
	hipDeviceSynchronize();

	
	float gpuTime_ms= 0;
	hipEventElapsedTime(&gpuTime_ms, start, stop);

	printf("runtime (ms) : %f\n", gpuTime_ms);

	// check data
	bool success = 1;
	for(int i=0; i< N * num_streams; i++) {
		if (abs(c_h[i] - 3.3f) > 1e-6) {
			fprintf(stderr, "%d : %f  (error)!\n", i, c_h[i]);
			success = 0;
			break;
		}
	}

	if(success) {
		printf("\nSuccess! Exit.\n");	
	}

	//------------------------------------------------------------------------//
	// free 
	//------------------------------------------------------------------------//

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

	hipHostFree(a_h);
	hipHostFree(b_h);
	hipHostFree(c_h);

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	hipDeviceReset();

	return 0;
}
